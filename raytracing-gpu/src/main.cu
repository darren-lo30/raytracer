#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include "utils/utils.h"
#include "render/ray_renderer.h"
#include "math/ray.h"
#include "math/vec3.h"
#include "render/camera.h"
#include "objects/hittable.h"
#include "objects/hittable_list.h"
#include "objects/sphere.h"
#include "materials/metal.h"
#include "materials/lambertian.h"
#include "materials/dielectric.h"
#include <hiprand/hiprand_kernel.h>
#include "display/window.h"
#include "render/shader.h"
#include "objects/triangle.h"
#include "objects/model_loader.h"
#include <time.h>
#include "render/window_renderer.h"
#include "utils/rand.h"
#include "objects/ray_model.h"

__global__ void random_scene(HittableList** world, RayModel *ray_model) {
  if(threadIdx.x != 0 || blockIdx.x != 0) return;

  const int size = 25 * 25 + 20;
  Hittable** objects = new Hittable*[size];
  *world = new HittableList(size, objects);

  hiprandState state_val;
  hiprand_init(1984, 0, 0, &state_val);
  hiprandState *state = &state_val;


  auto ground_material = new Lambertian(color(0.5, 0.5, 0.5));

  for (int a = -11; a < 11; a++) {
    for (int b = -11; b < 11; b++) {
      auto choose_mat = random_float(state);
      point3 center(a + 0.9*random_float(state), 0.2, b + 0.9*random_float(state));
      if ((center - point3(4, 0.2, 0)).length() > 0.9) {
        Material* sphere_mat;

        if (choose_mat < 0.8) {
          // diffuse
          auto albedo = random_vec3(state) * random_vec3(state);
          sphere_mat = new Lambertian(albedo);
        } else if (choose_mat < 0.95) {
          // metal
          auto albedo = random_vec3(state, 0.5, 1);
          auto fuzz = random_float(state, 0, 0.5);
          sphere_mat = new Metal(albedo, fuzz);
        } else {
          // glass
          sphere_mat = new Dielectric(1.5);
        }

        (*world)->add(new Sphere(center, 0.2, sphere_mat));
      }
    }
  }

  auto material1 = new Dielectric(1.5);
  (*world)->add(new Sphere(point3(0, 1, 0), 1.0, material1));

  auto material2 = new Lambertian(color(0.4, 0.2, 0.1));
  (*world)->add(new Sphere(point3(-4, 1, 0), 1.0, material2));

  auto material3 = new Metal(color(0.7, 0.6, 0.5), 0.0);
  (*world)->add(new Sphere(point3(4, 1, 0), 1.0, material3));

  (*world)->add(ray_model); 
  for(int i = 0; i<ray_model->meshes[0]->num_triangles; ++i) {
    ray_model->meshes[0]->triangles[i]->setMat(ground_material);
  }
  // (*world)->add(new Sphere(point3(0, 0, 3), 1, material3));
  // (*world)->add(new Triangle(point3(2,3,-1), point3(-5, 0, -1), point3(5, 0, -1), ground_material));
  // (*world)->add(new Sphere(point3(0, 0, -9), 3, ground_material));

}

int main() {
  // Metal test(color(3, 3, 3), 1.f);
  const float aspect_ratio = 16.0/9.0;
  RayRenderer renderer = RayRenderer(1000, aspect_ratio);

  point3 lookfrom(13,10,3);
  point3 lookat(0,0,0);
  vec3 vup(0,1,0);
  auto dist_to_focus = 10.0;
  auto aperture = 0.1;

  // Allocate camera
  Camera* camera = new Camera(lookfrom, lookat, vup, 20, aspect_ratio, aperture, dist_to_focus);

  Model model = ModelLoader::load_model("models/cube/cube.obj");
  auto ray_cube = from_model(model);
  checkCudaErrors(hipDeviceSynchronize());
    
  // Allocate world
  // Need to allocate a pointer so that we can create object with "new" in GPU so that virtual functions can run on device
  HittableList **world; 
  checkCudaErrors(hipMallocManaged(&world, sizeof(HittableList*))); 
  checkCudaErrors(hipDeviceSynchronize());
  random_scene<<<1, 1>>>(world, ray_cube);
  checkCudaErrors(hipDeviceSynchronize());

  printf("Done setting up scene\n");
  printf("Rendering...\n");
  clock_t start, stop;
  start = clock();

  color* fb = renderer.render_fb(camera, *world);

  stop = clock();
  double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
  std::cout << "Render took " << timer_seconds << " seconds.\n";

  Window window("render", (unsigned int) renderer.get_image_width(), (unsigned int) renderer.get_image_height());


  WindowRenderer scene_renderer = WindowRenderer();
  unsigned int scene_texture = WindowRenderer::gen_scene_texture(fb, renderer.get_image_width(), renderer.get_image_height());

  while(!glfwWindowShouldClose(window.get_id())) {
    glClearColor(1.0f, 0.3f, 0.3f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);

    scene_renderer.render_scene_to_window(scene_texture);
    
    glfwSwapBuffers(window.get_id());
    glfwPollEvents();
  }
}